#include "hip/hip_runtime.h"
#include "RayTriangleIntersection.h"
#include <iostream>

#define BLOCK_SIZE 256

// CUDA kernel for ray-triangle intersection
__global__ void rayTriangleIntersectKernel(
    Ray* rays,
    Triangle* triangles,
    Intersection* intersections,
    int numTests)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numTests) {
        const float EPSILON = 1e-8f;
        Ray ray = rays[idx];
        Triangle tri = triangles[idx];

        float3 edge1 = make_float3(
            tri.v1.x - tri.v0.x,
            tri.v1.y - tri.v0.y,
            tri.v1.z - tri.v0.z);

        float3 edge2 = make_float3(
            tri.v2.x - tri.v0.x,
            tri.v2.y - tri.v0.y,
            tri.v2.z - tri.v0.z);

        float3 h = cross(ray.direction, edge2);
        float a = dot(edge1, h);

        if (fabs(a) < EPSILON) {
            intersections[idx].hit = false; 
            return;
        }

        float f = 1.0f / a;
        float3 s = make_float3(
            ray.origin.x - tri.v0.x,
            ray.origin.y - tri.v0.y,
            ray.origin.z - tri.v0.z);

        float u = f * dot(s, h);
        if (u < 0.0f || u > 1.0f) {
            intersections[idx].hit = false;
            return;
        }

        float3 q = cross(s, edge1);
        float v = f * dot(ray.direction, q);
        if (v < 0.0f || u + v > 1.0f) {
            intersections[idx].hit = false;
            return;
        }

        float t = f * dot(edge2, q);
        if (t > EPSILON) { 
            intersections[idx].hit = true;
            intersections[idx].t = t;
            intersections[idx].u = u;
            intersections[idx].v = v;
        } else {
            intersections[idx].hit = false; 
        }
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
}

void performRayTriangleIntersections(
    Ray* rays,
    Triangle* triangles,
    Intersection* intersections,
    int numTests)
{
    Ray* d_rays = nullptr;
    Triangle* d_triangles = nullptr;
    Intersection* d_intersections = nullptr;

    size_t sizeRays = numTests * sizeof(Ray);
    size_t sizeTriangles = numTests * sizeof(Triangle);
    size_t sizeIntersections = numTests * sizeof(Intersection);

    checkCudaError(hipMalloc(&d_rays, sizeRays), "Allocating device memory for rays");
    checkCudaError(hipMalloc(&d_triangles, sizeTriangles), "Allocating device memory for triangles");
    checkCudaError(hipMalloc(&d_intersections, sizeIntersections), "Allocating device memory for intersections");

    checkCudaError(hipMemcpy(d_rays, rays, sizeRays, hipMemcpyHostToDevice), "Copying rays to device");
    checkCudaError(hipMemcpy(d_triangles, triangles, sizeTriangles, hipMemcpyHostToDevice), "Copying triangles to device");

    int numBlocks = (numTests + BLOCK_SIZE - 1) / BLOCK_SIZE;
    rayTriangleIntersectKernel<<<numBlocks, BLOCK_SIZE>>>(d_rays, d_triangles, d_intersections, numTests);
    checkCudaError(hipGetLastError(), "Launching kernel");

    checkCudaError(hipMemcpy(intersections, d_intersections, sizeIntersections, hipMemcpyDeviceToHost), "Copying intersections to host");

    hipFree(d_rays);
    hipFree(d_triangles);
    hipFree(d_intersections);
}
